#include "GPUGraphStore.cuh"
#include <algorithm>
#include <functional>
#include <iostream>
#include <hip/hip_runtime.h>

#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <unordered_map>
#include <numeric>
#include <math.h>
#include <thread>
#include <numeric>
#include <chrono>
#include <random>

#include <cstdint>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <time.h>
#include <sys/types.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <time.h>

void mmap_trainingset_read(std::string &training_file, std::vector<int32_t>& training_set_ids){
    int64_t t_idx = 0;
    int32_t fd = open(training_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<training_file<<"\n";
    }
    // int64_t buf_len = lseek(fd, 0, SEEK_END);
    int64_t buf_len = int64_t(int64_t(training_set_ids.size()) * 4); 
    const int32_t* buf = (int32_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int32_t* buf_end = buf + buf_len/sizeof(int32_t);
    int32_t temp;
    while(buf < buf_end){
        temp = *buf;
        training_set_ids[t_idx++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void mmap_partition_read(std::string &partition_file, int32_t* partition_index){
    int64_t part_idx = 0;
    int32_t fd = open(partition_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<partition_file<<"\n";
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const int32_t* buf = (int32_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int32_t* buf_end = buf + buf_len/sizeof(int32_t);
    int32_t temp;
    while(buf < buf_end){
        temp = *buf;
        partition_index[part_idx++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void mmap_indptr_read(std::string &indptr_file, int64_t* indptr){
    int64_t indptr_index = 0;
    int32_t fd = open(indptr_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<indptr_file<<"\n";
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const int64_t *buf = (int64_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int64_t* buf_end = buf + buf_len/sizeof(int64_t);
    int64_t temp;
    while(buf < buf_end){
        temp = *buf;
        indptr[indptr_index++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void mmap_indices_read(std::string &indices_file, int32_t* indices){
    int64_t indices_index = 0;
    int32_t fd = open(indices_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<indices_file<<"\n";
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const int32_t *buf = (int32_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int32_t* buf_end = buf + buf_len/sizeof(int32_t);
    int32_t temp;
    while(buf < buf_end){
        temp = *buf;
        indices[indices_index++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void mmap_features_read(std::string &features_file, float* features){
    int64_t n_idx = 0;
    int32_t fd = open(features_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<features_file<<"\n";
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const float *buf = (float *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const float* buf_end = buf + buf_len/sizeof(float);
    float temp;
    while(buf < buf_end){
        temp = *buf;
        features[n_idx++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void mmap_labels_read(std::string &labels_file, std::vector<int32_t>& labels){
    int64_t n_idx = 0;
    int32_t fd = open(labels_file.c_str(), O_RDONLY);
    if(fd == -1){
        std::cout<<"cannout open file: "<<labels_file<<"\n";
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const int32_t *buf = (int32_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int32_t* buf_end = buf + buf_len/sizeof(int32_t);
    int32_t temp;
    while(buf < buf_end){
        temp = *buf;
        labels[n_idx++] = temp;
        buf++;
    }
    close(fd);
    return;
}

void GPUGraphStore::EnableP2PAccess(){
    int32_t central_device = -1;
    hipGetDevice(&central_device);

    int32_t device_count = -1;
    hipGetDeviceCount(&device_count);
    for(int32_t i = 0; i < device_count; i++){
        hipSetDevice(i);
        cudaCheckError();
        for(int32_t j = 0; j < device_count; j++){
          if(j != i){
            int32_t accessible = 0;
            hipDeviceCanAccessPeer(&accessible, i, j);
            cudaCheckError();
            if(accessible){
              hipDeviceEnablePeerAccess(j, 0);
              cudaCheckError();
            }
          }
        }
      }
    hipSetDevice(central_device);
    central_device_ = central_device;
}

void GPUGraphStore::ConfigPartition(BuildInfo* info, int32_t shard_count){

    shard_to_device_.resize(shard_count);
    for(int32_t i = 0; i < shard_count; i++){
        shard_to_device_[i] = i;
    }
    shard_to_partition_.resize(shard_count);
    for(int32_t i = 0; i < shard_count; i++){
        shard_to_partition_[i] = i;
    }
    for(int32_t i = 0; i < shard_count; i++){
        info->shard_to_partition.push_back(shard_to_partition_[i]);
    }
    for(int32_t i = 0; i < shard_count; i++){
        info->shard_to_device.push_back(shard_to_device_[i]);
    }

    info->partition_count = shard_count;
}

void GPUGraphStore::ReadMetaFIle(BuildInfo* info){
    std::istringstream iss;
    std::string buff;
    std::ifstream Metafile("./meta_config");
    if(!Metafile.is_open()){
     std::cout<<"unable to open meta config file"<<"\n";
    }
    getline(Metafile, buff);
    iss.clear();
    iss.str(buff);
    iss >> dataset_path_;
    std::cout<<"Dataset path:       "<<dataset_path_<<"\n";
    iss >> raw_batch_size_;
    std::cout<<"Raw Batchsize:      "<<raw_batch_size_<<"\n";
    info->raw_batch_size = raw_batch_size_;
    iss >> node_num_;
    std::cout<<"Graph nodes num:    "<<node_num_<<"\n";
    iss >> edge_num_;
    std::cout<<"Graph edges num:    "<<edge_num_<<"\n";
    iss >> cache_edge_num_;
    std::cout<<"Cache edges num:    "<<cache_edge_num_<<"\n";
    iss >> float_attr_len_;
    std::cout<<"Feature dim:        "<<float_attr_len_<<"\n";
    iss >> training_set_num_;
    std::cout<<"Training set num:   "<<training_set_num_<<"\n";
    iss >> validation_set_num_;
    std::cout<<"Validation set num: "<<validation_set_num_<<"\n";
    iss >> testing_set_num_;
    std::cout<<"Testing set num:    "<<testing_set_num_<<"\n";
    iss >> cache_cap_;
    std::cout<<"Cache capacity:     "<<cache_cap_<<"\n";
    iss >> cache_way_;
    std::cout<<"Cache way num:      "<<cache_way_<<"\n"; 
    iss >> future_batch_;
    std::cout<<"Predict by K batch: "<<future_batch_<<"\n"; 
    iss >> epoch_;
    std::cout<<"Train epoch:        "<<epoch_<<"\n";
    int numPages;
    iss >> numPages;
    std::cout<<"Num of pages:       "<<numPages<<"\n";
    int pageSize;
    iss >> pageSize;
    std::cout<<"Page size:          "<<pageSize<<"\n";

    info->epoch = epoch_;

    info->cudaDevice = 0;
    info->cudaDeviceId = 0;
    info->blockDevicePath = nullptr;
    info->controllerPath = nullptr;
    info->controllerId = 0;
    info->adapter = 0;
    info->segmentId = 0;
    info->nvmNamespace = 1;
    info->doubleBuffered = false;
    info->numReqs = 1;
    info->numPages = numPages;
    info->startBlock = 0;
    info->stats = false;
    info->output = nullptr;
    info->numThreads = 64;
    info->blkSize = 64;
    info->domain = 0;
    info->bus = 0;
    info->devfn = 0;
    info->n_ctrls = 12;
    info->queueDepth = 1024;
    info->numQueues = 256;
    info->pageSize = pageSize;
    info->numElems = int64_t(node_num_) * float_attr_len_;
    info->random = true;
    info->ssdtype = 0;

}

void GPUGraphStore::Load_Graph(BuildInfo* info){
    std::cout<<"Start load graph\n";

    // int32_t partition_count = info->partition_count;
    int32_t node_num = node_num_;
    int64_t edge_num = edge_num_;
    info->total_edge_num = edge_num;
    info->cache_edge_num = cache_edge_num_;

    //uva
    hipHostAlloc(&(info->csr_node_index), int64_t(int64_t(node_num + 1)*sizeof(int64_t)), hipHostMallocMapped);
    hipHostAlloc(&(info->csr_dst_node_ids), int64_t(int64_t(edge_num) * sizeof(int32_t)), hipHostMallocMapped);
    std::string edge_src_path = dataset_path_ + "edge_src";
    std::string edge_dst_path = dataset_path_ + "edge_dst";

    mmap_indptr_read(edge_src_path, info->csr_node_index);
    mmap_indices_read(edge_dst_path, info->csr_dst_node_ids);
}


void GPUGraphStore::Load_Feature(BuildInfo* info){
    std::cout<<"start load node\n";

    int32_t partition_count = info->partition_count;

    int32_t node_num = node_num_;
    int32_t nf = float_attr_len_;

    info->numElems = uint64_t(node_num) * nf;

    (info->training_set_ids).resize(partition_count);
    (info->training_labels).resize(partition_count);
    (info->validation_set_ids).resize(partition_count);
    (info->validation_labels).resize(partition_count);
    (info->testing_set_ids).resize(partition_count);
    (info->testing_labels).resize(partition_count);

    std::string training_path = dataset_path_  + "trainingset";
    std::string validation_path = dataset_path_  + "validationset";
    std::string testing_path = dataset_path_  + "testingset";
    // std::string training_path = dataset_path_  + "train_ids";
    // std::string validation_path = dataset_path_  + "valid_ids";
    // std::string testing_path = dataset_path_  + "test_ids";
    std::string features_path = dataset_path_ + "features";
    std::string labels_path = dataset_path_ + "labels";
    // std::string labels_path = dataset_path_ + "labels_raw";

    std::string partition_path = dataset_path_ + "partition_" + std::to_string(partition_count) + "_bn";

    std::vector<int32_t> training_ids;
    training_ids.resize(training_set_num_);
    std::vector<int32_t> validation_ids;
    validation_ids.resize(validation_set_num_);
    std::vector<int32_t> testing_ids;
    testing_ids.resize(testing_set_num_);
    std::vector<int32_t> all_labels;
    all_labels.resize(node_num);
    // std::vector<char> partition_index;
    int32_t* partition_index = (int32_t*)malloc(int64_t(node_num) * sizeof(int32_t));
    // partition_index.resize(node_num);
    float* host_float_attrs;
    hipHostAlloc(&host_float_attrs, int64_t(int64_t(int64_t(node_num) * nf) * sizeof(float)), hipHostMallocMapped);
    cudaCheckError();


    mmap_trainingset_read(training_path, training_ids);
    mmap_trainingset_read(validation_path, validation_ids);
    mmap_trainingset_read(testing_path, testing_ids);
    // mmap_features_read(features_path, host_float_attrs);
    // mmap_labels_read(labels_path, all_labels);
    mmap_partition_read(partition_path, partition_index);

    std::cout<<"Finish Reading All Files\n";
    // partition nodes

    std::cout<<training_set_num_<<"\n";
    int trainingset_count = 0;
    std::cout<<"partition count "<<partition_count<<"\n";
    for(int32_t i = 0; i < training_set_num_; i+=1){
        int32_t tid = training_ids[i];
        int32_t part_id = tid % partition_count;
        // int32_t part_id = partition_index[tid];
        // part_id = (part_id / 2) * 2 + (tid % 2);
        if(part_id < partition_count){
            (info->training_set_ids[part_id]).push_back(tid);
            trainingset_count ++ ;
            // (info->training_set_ids[part_id]).push_back(training_ids[i + 1]);
            // (info->training_set_ids[part_id]).push_back(training_ids[i + 2]);
        }
        // if(part_id == 5){
        //     part_id = 7;
        // }else if(part_id == 7){
        //     part_id = 5;
        // }else if(part_id == 1){
        //     part_id = 3;
        // }else if(part_id == 3){
        //     part_id = 1;
        // }else if(part_id == 0){
        //     part_id = 4;
        // }else if(part_id == 4){
        //     part_id = 0;
        // }
        // part_id = (part_id / 4) * 4 + (tid % 4);
        // 


        // if(part_id < partition_count / 2){
        //     part_id = tid % (partition_count / 2);
        // }else{
        //     part_id = (partition_count / 2) + (tid % (partition_count / 2));
        // }




    }
    std::cout<<"training set count "<<trainingset_count<<"\n";

    for(int32_t i = 0; i < validation_set_num_; i++){
        int32_t tid = validation_ids[i];
        int32_t part_id = tid % partition_count;
        // int32_t part_id = partition_index[tid];
        // if(part_id < partition_count / 2){
        //     part_id = tid % (partition_count / 2);
        // }else{
        //     part_id = (partition_count / 2) + (tid % (partition_count / 2));
        // }

        if(part_id < partition_count){
            (info->validation_set_ids[part_id]).push_back(tid);
        }
    }

    for(int32_t i = 0; i < testing_set_num_; i++){
        int32_t tid = testing_ids[i];
        int32_t part_id = tid % partition_count;
        // int32_t part_id = partition_index[tid];
        // if(part_id < partition_count / 2){
        //     part_id = tid % (partition_count / 2);
        // }else{
        //     part_id = (partition_count / 2) + (tid % (partition_count / 2));
        // }
        
        if(part_id < partition_count){
            (info->testing_set_ids[part_id]).push_back(tid);
        }
    }
    free(partition_index);

    //partition labels
    for(int32_t part_id = 0; part_id < partition_count; part_id++){
        for(int32_t i = 0; i < info->training_set_ids[part_id].size(); i++){
            int32_t ts_label = 0;//all_labels[info->training_set_ids[part_id][i]];
            info->training_labels[part_id].push_back(ts_label);
        }
        // info->training_labels[part_id].resize(info->training_set_ids[part_id].size());
        info->training_set_num.push_back(info->training_set_ids[part_id].size());
    }
    std::cout<<info->training_set_num[0]<<" "<<info->training_set_ids[0].size()<<"\n";
    for(int32_t part_id = 0; part_id < partition_count; part_id++){
        for(int32_t i = 0; i < info->validation_set_ids[part_id].size(); i++){
            int32_t ts_label = 0;//all_labels[info->validation_set_ids[part_id][i]];
            info->validation_labels[part_id].push_back(ts_label);
        }
        // info->validation_labels[part_id].resize(info->validation_set_ids[part_id].size());
        info->validation_set_num.push_back(info->validation_set_ids[part_id].size());
    }

    for(int32_t part_id = 0; part_id < partition_count; part_id++){
        for(int32_t i = 0; i < info->testing_set_ids[part_id].size(); i++){
            int32_t ts_label = 0;//all_labels[info->testing_set_ids[part_id][i]];
            info->testing_labels[part_id].push_back(ts_label);
        }
        // info->testing_labels.resize(info->testing_set_ids[part_id].size());
        info->testing_set_num.push_back(info->testing_set_ids[part_id].size());
    }

    info->host_float_attrs = host_float_attrs;
    info->float_attr_len = float_attr_len_;
    info->host_int_attrs = nullptr;
    info->int_attr_len = 0;
    info->total_num_nodes = node_num_;
    std::cout<<"Finish Partition\n";

    // training_ids.clear();
    // validation_ids.clear();
    // testing_ids.clear();
    // all_labels.clear();
}

void GPUGraphStore::Initialze(int32_t shard_count){

    BuildInfo* info = new BuildInfo();

    EnableP2PAccess();
    
    ConfigPartition(info, shard_count);

    ReadMetaFIle(info);

    Load_Graph(info);

    Load_Feature(info);

    env_ = NewIPCEnv(shard_count);
    env_ -> Coordinate(info);

    node_ = NewGPUMemoryNodeStorage();
    node_ -> Build(info);  

    graph_ = NewGPUMemoryGraphStorage();
    graph_ -> Build(info);

    cudaCheckError();

    cache_ = new GPUCache();
    std::vector<int> device;

    for(int32_t i = 0; i < shard_to_device_.size(); i++){
        if(shard_to_device_[i] >= 0){
            device.push_back(shard_to_device_[i]);
        }
    }

    int32_t train_step = env_->GetTrainStep();

    hipSetDevice(0);
    cache_ -> Initialize(device, cache_cap_, 0, float_attr_len_, future_batch_, cache_way_, train_step);
    hipSetDevice(0);
    std::cout<<"Storage Initialized\n";
}

GPUGraphStorage* GPUGraphStore::GetGraph(){
    return graph_;
}

GPUNodeStorage* GPUGraphStore::GetNode(){
    return node_;
}

GPUCache* GPUGraphStore::GetCache(){
    return cache_;
}

IPCEnv* GPUGraphStore::GetIPCEnv(){
    return env_;
}

int32_t GPUGraphStore::Shard_To_Device(int32_t shard_id){
    return shard_to_device_[shard_id];
}

int32_t GPUGraphStore::Shard_To_Partition(int32_t shard_id){
    return shard_to_partition_[shard_id];
}

int32_t GPUGraphStore::Central_Device(){
    return central_device_;
}