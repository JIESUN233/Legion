#include "GPU_Node_Storage.cuh"
#include <iostream>

class GPUMemoryNodeStorage : public GPUNodeStorage{
public: 
    GPUMemoryNodeStorage(){
    }

    virtual ~GPUMemoryNodeStorage(){};

    void Build(BuildInfo* info) override {
        int32_t partition_count = info->partition_count;
        total_num_nodes_ = info->total_num_nodes;
        int_attr_len_ = info->int_attr_len;
        float_attr_len_ = info->float_attr_len;
        int64_t* host_int_attrs = info->host_int_attrs;
        float* host_float_attrs = info->host_float_attrs;

        if(int_attr_len_ > 0){
            hipHostGetDevicePointer(&int_attrs_, host_int_attrs, 0);
        }
        if(float_attr_len_ > 0){
            hipHostGetDevicePointer(&float_attrs_, host_float_attrs, 0);
        }
        cudaCheckError();

        hipSetDevice(0);

        training_set_num_.resize(partition_count);
        training_set_ids_.resize(partition_count);
        training_labels_.resize(partition_count);

        validation_set_num_.resize(partition_count);
        validation_set_ids_.resize(partition_count);
        validation_labels_.resize(partition_count);

        testing_set_num_.resize(partition_count);
        testing_set_ids_.resize(partition_count);
        testing_labels_.resize(partition_count);

        partition_count_ = partition_count;

        for(int32_t i = 0; i < info->shard_to_partition.size(); i++){
            int32_t part_id = info->shard_to_partition[i];
            int32_t device_id = info->shard_to_device[i];
            /*part id = 0, 1, 2...*/

            training_set_num_[part_id] = info->training_set_num[part_id];
            // std::cout<<"Training set count "<<training_set_num_[part_id]<<" "<<info->training_set_num[part_id]<<"\n";

            validation_set_num_[part_id] = info->validation_set_num[part_id];
            testing_set_num_[part_id] = info->testing_set_num[part_id];

            hipSetDevice(device_id);
            cudaCheckError();

            // std::cout<<"Training set on device "<<part_id<<" "<<training_set_num_[part_id]<<"\n";
            // std::cout<<"Testing set on device "<<part_id<<" "<<testing_set_num_[part_id]<<"\n";

            int32_t* train_ids;
            hipMalloc(&train_ids, training_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(train_ids, info->training_set_ids[part_id].data(), training_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            training_set_ids_[part_id] = train_ids;
            cudaCheckError();

            int32_t* valid_ids;
            hipMalloc(&valid_ids, validation_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(valid_ids, info->validation_set_ids[part_id].data(), validation_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            validation_set_ids_[part_id] = valid_ids;
            cudaCheckError();

            int32_t* test_ids;
            hipMalloc(&test_ids, testing_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(test_ids, info->testing_set_ids[part_id].data(), testing_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            testing_set_ids_[part_id] = test_ids;
            cudaCheckError();

            int32_t* train_labels;
            hipMalloc(&train_labels, training_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(train_labels, info->training_labels[part_id].data(), training_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            training_labels_[part_id] = train_labels;
            cudaCheckError();

            int32_t* valid_labels;
            hipMalloc(&valid_labels, validation_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(valid_labels, info->validation_labels[part_id].data(), validation_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            validation_labels_[part_id] = valid_labels;
            cudaCheckError();

            int32_t* test_labels;
            hipMalloc(&test_labels, testing_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(test_labels, info->testing_labels[part_id].data(), testing_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            testing_labels_[part_id] = test_labels;
            cudaCheckError();

        }

        hipMalloc(&d_req_count_, sizeof(unsigned long long));
        hipMemset(d_req_count_, 0, sizeof(unsigned long long));
        cudaCheckError();

    };

    void Finalize() override {
        hipHostFree(float_attrs_);
        for(int32_t i = 0; i < partition_count_; i++){
            hipSetDevice(i);
            hipFree(training_set_ids_[i]);
            hipFree(validation_set_ids_[i]);
            hipFree(testing_set_ids_[i]);
            hipFree(training_labels_[i]);
            hipFree(validation_labels_[i]);
            hipFree(testing_labels_[i]);
        }
    }

    int32_t* GetTrainingSetIds(int32_t part_id) const override {
        return training_set_ids_[part_id];
    }
    int32_t* GetValidationSetIds(int32_t part_id) const override {
        return validation_set_ids_[part_id];
    }
    int32_t* GetTestingSetIds(int32_t part_id) const override {
        return testing_set_ids_[part_id];
    }

	int32_t* GetTrainingLabels(int32_t part_id) const override {
        return training_labels_[part_id];
    };
    int32_t* GetValidationLabels(int32_t part_id) const override {
        return validation_labels_[part_id];
    }
    int32_t* GetTestingLabels(int32_t part_id) const override {
        return testing_labels_[part_id];
    }

    int32_t TrainingSetSize(int32_t part_id) const override {
        return training_set_num_[part_id];
    }
    int32_t ValidationSetSize(int32_t part_id) const override {
        return validation_set_num_[part_id];
    }
    int32_t TestingSetSize(int32_t part_id) const override {
        return testing_set_num_[part_id];
    }

    int32_t TotalNodeNum() const override {
        return total_num_nodes_;
    }
	int64_t* GetAllIntAttr() const override {
        return int_attrs_;
    }
    int32_t GetIntAttrLen() const override {
        return int_attr_len_;
    }
    float* GetAllFloatAttr() const override {
        return float_attrs_;
    }
    int32_t GetFloatAttrLen() const override {
        return float_attr_len_;
    }

    void Print(BuildInfo* info) override {
    }

    void GetBamFloatAttr(float** cache_float_attrs, int32_t float_attr_len,
                        int32_t* sampled_ids, int32_t* cache_index, int32_t cache_capacity,
                        int32_t* node_counter, float* dst_float_buffer,
                        int32_t total_num_nodes,
                        int32_t dev_id,
                        int32_t op_id, hipStream_t strm_hdl) override {

    }



private:
    std::vector<int> training_set_num_;
    std::vector<int> validation_set_num_;
    std::vector<int> testing_set_num_;

    std::vector<int32_t*> training_set_ids_;
    std::vector<int32_t*> validation_set_ids_;
    std::vector<int32_t*> testing_set_ids_;

    std::vector<int32_t*> training_labels_;
    std::vector<int32_t*> validation_labels_;
    std::vector<int32_t*> testing_labels_;

    int32_t partition_count_;
    int32_t total_num_nodes_;
    int64_t* int_attrs_;
    int32_t int_attr_len_;
    float* float_attrs_;
    int32_t float_attr_len_;

    unsigned long long* d_req_count_;


    friend GPUNodeStorage* NewGPUMemoryNodeStorage();
};

extern "C" 
GPUNodeStorage* NewGPUMemoryNodeStorage(){
    GPUMemoryNodeStorage* ret = new GPUMemoryNodeStorage();
    return ret;
}
