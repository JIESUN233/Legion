#include "hip/hip_runtime.h"
#include "GPU_Graph_Storage.cuh"
#include <iostream>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

__global__ void assign_memory(int32_t** int32_pptr, int32_t* int32_ptr, int64_t** int64_pptr, int64_t* int64_ptr, int32_t device_id){
    int32_pptr[device_id] = int32_ptr;
    int64_pptr[device_id] = int64_ptr;
}


__global__ void GetNeighborCount(int32_t* QT, int32_t Kg, int32_t Ki, int32_t capacity, int64_t* csr_node_index_cpu, int64_t* neighbor_count){
    for(int32_t thread_idx = threadIdx.x + blockDim.x * blockIdx.x; thread_idx < capacity; thread_idx += gridDim.x * blockDim.x){
        int32_t cache_id = QT[thread_idx * Kg + Ki];
        int64_t count = csr_node_index_cpu[cache_id + 1] - csr_node_index_cpu[cache_id];
        neighbor_count[thread_idx] = count;
    }
}

__global__ void TopoFillUp(int32_t* QT, int32_t Kg, int32_t Ki, int32_t capacity, 
                            int64_t* csr_node_index_cpu, int32_t* csr_dst_node_ids_cpu, 
                             int64_t* d_csr_node_index, int32_t* d_csr_dst_node_ids){
    for(int32_t thread_idx = threadIdx.x + blockDim.x * blockIdx.x; thread_idx < capacity; thread_idx += gridDim.x * blockDim.x){
        int32_t cache_id = QT[thread_idx * Kg + Ki];
        int64_t count = csr_node_index_cpu[cache_id + 1] - csr_node_index_cpu[cache_id];
        for(int i = 0; i < count; i++){
            int32_t neighbor_id = csr_dst_node_ids_cpu[csr_node_index_cpu[cache_id] + i];
            int64_t start_off = d_csr_node_index[thread_idx];
            d_csr_dst_node_ids[start_off + i] = neighbor_id;
        }
    }
}

/*in this version, partition id = shard id = device id*/
class GPUMemoryGraphStorage : public GPUGraphStorage {
public:
    GPUMemoryGraphStorage() {
    }

    virtual ~GPUMemoryGraphStorage() {
    }

    void Build(BuildInfo* info) override {
        int32_t partition_count = info->partition_count;
        partition_count_ = partition_count;
        node_num_ = info->total_num_nodes;
        edge_num_ = info->total_edge_num;
        cache_edge_num_ = info->cache_edge_num;

        // shard count == partition count now
        csr_node_index_.resize(partition_count_);
        csr_dst_node_ids_.resize(partition_count_);
        partition_index_.resize(partition_count_);
        partition_offset_.resize(partition_count_);

        d_global_count_.resize(partition_count);
        h_global_count_.resize(partition_count);
        h_cache_hit_.resize(partition_count);
        find_iter_.resize(partition_count);
        h_batch_size_.resize(partition_count);

        for(int32_t i = 0; i < partition_count; i++){
            hipSetDevice(i);
            hipMalloc(&csr_node_index_[i], (partition_count + 1) * sizeof(int64_t*));
            hipMalloc(&csr_dst_node_ids_[i], (partition_count + 1) * sizeof(int32_t*));
            hipMalloc(&d_global_count_[i], 4);
            h_global_count_[i] = (int32_t*)malloc(4);
            h_cache_hit_[i] = 0;
            find_iter_[i] = 0;
            h_batch_size_[i] = 0;
        }

        src_size_.resize(partition_count);
        dst_size_.resize(partition_count);
        cudaCheckError();

        hipSetDevice(0);

        int64_t* pin_csr_node_index;
        int32_t* pin_csr_dst_node_ids;

        h_csr_node_index_ = info->csr_node_index;
        h_csr_dst_node_ids_ = info->csr_dst_node_ids;
        
        hipHostGetDevicePointer(&pin_csr_node_index, h_csr_node_index_, 0);
        hipHostGetDevicePointer(&pin_csr_dst_node_ids, h_csr_dst_node_ids_, 0);
        assign_memory<<<1,1>>>(csr_dst_node_ids_[0], pin_csr_dst_node_ids, csr_node_index_[0], pin_csr_node_index, partition_count);
        cudaCheckError();

        csr_node_index_cpu_ = pin_csr_node_index;
        csr_dst_node_ids_cpu_ = pin_csr_dst_node_ids;
        
    }
    

    void GraphCache(int32_t* QT, int32_t Ki, int32_t Kg, int32_t capacity){
        hipMemcpy(csr_node_index_[Ki * Kg], csr_node_index_[0], (partition_count_ + 1) * sizeof(int64_t*), hipMemcpyDeviceToDevice);
        cudaCheckError();
        hipMemcpy(csr_dst_node_ids_[Ki * Kg], csr_dst_node_ids_[0], (partition_count_ + 1) * sizeof(int32_t*), hipMemcpyDeviceToDevice);
        cudaCheckError();
        for(int32_t i = 0; i < Kg; i++){
            hipSetDevice(Ki * Kg + i);
            int64_t* neighbor_count;
            hipMalloc(&neighbor_count, capacity * sizeof(int64_t));
            GetNeighborCount<<<80, 1024>>>(QT, Kg, i, capacity, csr_node_index_cpu_, neighbor_count);

            int64_t* d_csr_node_index;
            hipMalloc(&d_csr_node_index, (int64_t(capacity + 1)*sizeof(int64_t)));
            hipMemset(d_csr_node_index, 0, (int64_t(capacity + 1)*sizeof(int64_t)));
            thrust::inclusive_scan(thrust::device, neighbor_count, neighbor_count + capacity, d_csr_node_index + 1);
            cudaCheckError();
            int64_t* h_csr_node_index = (int64_t*)malloc((capacity + 1) * sizeof(int64_t));
            hipMemcpy(h_csr_node_index, d_csr_node_index, (capacity + 1) * sizeof(int64_t), hipMemcpyDeviceToHost);
            
            int32_t* d_csr_dst_node_ids;
            hipMalloc(&d_csr_dst_node_ids, int64_t(int64_t(h_csr_node_index[capacity]) * sizeof(int32_t)));

            TopoFillUp<<<80, 1024>>>(QT, Kg, i, capacity, csr_node_index_cpu_, csr_dst_node_ids_cpu_, d_csr_node_index, d_csr_dst_node_ids);
            cudaCheckError();
    
            assign_memory<<<1,1>>>(csr_dst_node_ids_[Ki * Kg], d_csr_dst_node_ids, csr_node_index_[Ki * Kg], d_csr_node_index, Ki * Kg + i);
            cudaCheckError();
            hipFree(neighbor_count);
        }
        for(int32_t i = 1; i < Kg; i++){
            hipMemcpy(csr_node_index_[Ki * Kg + i], csr_node_index_[Ki * Kg], (partition_count_ + 1) * sizeof(int64_t*), hipMemcpyDeviceToDevice);
            cudaCheckError();
            hipMemcpy(csr_dst_node_ids_[Ki * Kg + i], csr_dst_node_ids_[Ki * Kg], (partition_count_ + 1) * sizeof(int32_t*), hipMemcpyDeviceToDevice);
            cudaCheckError();
        }
    }

    void Finalize() override {
        hipHostFree(csr_node_index_cpu_);
        hipHostFree(csr_dst_node_ids_cpu_);
        // for(int32_t i = 0; i < partition_count_; i++){
        //     hipFree(partition_index_[i]);
        //     hipFree(partition_offset_[i]);
        // }
    }

    //CSR
    int32_t GetPartitionCount() const override {
        return partition_count_;
    }
	int64_t** GetCSRNodeIndex(int32_t dev_id) const override {
		return csr_node_index_[dev_id];
	}
	int32_t** GetCSRNodeMatrix(int32_t dev_id) const override {
        return csr_dst_node_ids_[dev_id];
    }
    
    int64_t* GetCSRNodeIndexCPU() const override {
        return csr_node_index_cpu_;
    }

    int32_t* GetCSRNodeMatrixCPU() const override {
        return csr_dst_node_ids_cpu_;
    }

    int64_t Src_Size(int32_t part_id) const override {
        return src_size_[part_id];
    }
    int64_t Dst_Size(int32_t part_id) const override {
        return dst_size_[part_id];
    }
    char* PartitionIndex(int32_t dev_id) const override {
        return partition_index_[dev_id];
    }
    int32_t* PartitionOffset(int32_t dev_id) const override {
        return partition_offset_[dev_id];
    }

private:
    std::vector<int64_t> src_size_;	
	std::vector<int64_t> dst_size_;

    int32_t node_num_;
    int64_t edge_num_;
    int64_t cache_edge_num_;

	//CSR graph, every partition has a ptr copy
    int32_t partition_count_;
	std::vector<int64_t**> csr_node_index_;
	std::vector<int32_t**> csr_dst_node_ids_;	
    int64_t* csr_node_index_cpu_;
    int32_t* csr_dst_node_ids_cpu_;

    int64_t* h_csr_node_index_;
    int32_t* h_csr_dst_node_ids_;

    std::vector<char*> partition_index_;
    std::vector<int32_t*> partition_offset_;

    std::vector<int32_t*> h_global_count_;
    std::vector<int32_t*> d_global_count_;


    std::vector<int32_t> find_iter_;
    std::vector<int32_t> h_cache_hit_;
    std::vector<int32_t> h_batch_size_;
};

extern "C" 
GPUGraphStorage* NewGPUMemoryGraphStorage(){
    GPUMemoryGraphStorage* ret = new GPUMemoryGraphStorage();
    return ret;
}
